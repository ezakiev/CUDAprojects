﻿#include "hip/hip_runtime.h"

#include "cmath"
#include <stdio.h>

#define N 10 //количество элементов массива

// Ядро
__global__ void ProdD(double* a, double* b, double* c)
{
	int i = threadIdx.x; //индексирование
	if (i > N - 1) return; 	//проверка на выход за пределы массива
	//поэлементное умножение массивов
	c[i] = __dmul_rn(a[i], b[i]);
}

__global__ void ProdF(float* a, float* b, float* c)
{
	int i = threadIdx.x; //индексирование
	if (i > N - 1) return; 	//проверка на выход за пределы массива
	//поэлементное умножение массивов
	c[i] = __fmul_rn(a[i], b[i]);
}

int main()
{
	//переменные для замера времени работы
	hipEvent_t start_f, stop_f, start_d, stop_d;
	hipEventCreate(&start_f);
	hipEventCreate(&stop_f);
	hipEventCreate(&start_d);
	hipEventCreate(&stop_d);

	// выделение памяти под массивы на CPU
	float a_f[N], b_f[N], c_f[N];
	double a_d[N], b_d[N], c_d[N];
	// выделение памяти под массивы для копирования
	// на GPU
	float *dev_a_f, *dev_b_f, *dev_c_f;
	double *dev_a_d, *dev_b_d, *dev_c_d;

	// заполнение массивов
	for (int i = 0; i < N; i++)
	{
		a_f[i] = a_d[i] = i + 1;
		b_f[i] = b_d[i] = -2;
	}

	// выделение памяти под массивы на GPU
	hipMalloc((void**)&dev_a_f, N * sizeof(float));
	hipMalloc((void**)&dev_b_f, N * sizeof(float));
	hipMalloc((void**)&dev_c_f, N * sizeof(float));
	hipMalloc((void**)&dev_a_d, N * sizeof(double));
	hipMalloc((void**)&dev_b_d, N * sizeof(double));
	hipMalloc((void**)&dev_c_d, N * sizeof(double));

	// копирование данных в память GPU
	hipMemcpy(dev_a_f, a_f, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b_f, b_f, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a_d, a_d, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_b_d, b_d, N * sizeof(double), hipMemcpyHostToDevice);

	hipEventRecord(start_f, 0); //начало замера ядра float
	ProdF <<<1, N >>> (dev_a_f, dev_b_f, dev_c_f);
	hipEventRecord(stop_f, 0); //конец замера
	hipEventSynchronize(stop_f);

	float kernelTime_f;
	hipEventElapsedTime(&kernelTime_f, start_f, stop_f);
	printf("Float kernel time = %f ms\n", kernelTime_f);

	hipEventRecord(start_d, 0); //начало замера ядра double
	ProdD <<<1, N >>> (dev_a_d, dev_b_d, dev_c_d);
	hipEventRecord(stop_d, 0); //конец замера
	hipEventSynchronize(stop_d);

	float kernelTime_d;
	hipEventElapsedTime(&kernelTime_d, start_d, stop_d);
	printf("Double kernel time = %f ms\n", kernelTime_d);

	hipMemcpy(c_f, dev_c_f, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(c_d, dev_c_d, N * sizeof(double), hipMemcpyDeviceToHost);

	//переменные для результата
	float prod_f = 0;
	double prod_d = 0;

	//подсчёт скалярного произведения
	for (int i = 0; i < N; i++)
	{
		prod_f += c_f[i];
		prod_d += c_d[i];
	}

	//вывод результата
	printf("prod_f = %f\nprod_d = %f\n", prod_f, prod_d);

	hipFree(dev_a_f);
	hipFree(dev_b_f);
	hipFree(dev_c_f);
	hipFree(dev_a_d);
	hipFree(dev_b_d);
	hipFree(dev_c_d);
	return 0;
}