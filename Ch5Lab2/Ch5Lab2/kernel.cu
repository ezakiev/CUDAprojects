﻿#include "hip/hip_runtime.h"

#include "cmath"
#include <stdio.h>

#define N 10 //количество элементов массива

// Ядро
__global__ void Prod(float *a, float *b, float *c)
{
	int i = threadIdx.x; //индексирование
	if (i > N - 1) return; 	//проверка на выход за пределы массива
	//поэлементное умножение массивов
	c[i] = __fmul_rn(a[i], b[i]);
}

int main() 
{
	// выделение памяти под массивы на CPU
	float a[N], b[N], c[N];
	// выделение памяти под массивы для копирования
	// на GPU
	float *dev_a, *dev_b, *dev_c;

	// заполнение массивов
	for (int i = 0; i < N; i++)
	{
		a[i] = i + 1;
		b[i] = -2;
	}

	// выделение памяти под массивы на GPU
	hipMalloc((void**)&dev_a, N * sizeof(float));
	hipMalloc((void**)&dev_b, N * sizeof(float));
	hipMalloc((void**)&dev_c, N * sizeof(float));

	// копирование данных в память GPU
	hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

	Prod <<<1, N >>>(dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);

	float prod = 0; //переменная для результата

	//подсчёт скалярного произведения
	for (int i = 0; i < N; i++)
	{
		prod += c[i];
	}

	//вывод результата
	printf("prod = %f\n", prod);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}