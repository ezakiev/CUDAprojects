﻿#include "hip/hip_runtime.h"

#include "cmath"
#include <stdio.h>

#define N 10 //количество элементов массива

// Ядро
__global__ void Prod(int *a, int *b, int *c)
{
	int i = threadIdx.x; //индексирование
	if (i > N - 1) return; 	//проверка на выход за пределы массива
	//поэлементное умножение массивов
	c[i] = a[i] * b[i];
}

int main() 
{
	// выделение памяти под массивы на CPU
	int a[N], b[N], c[N];
	// выделение памяти под массивы для копирования
	// на GPU
	int *dev_a, *dev_b, *dev_c;

	// заполнение массивов
	for (int i = 0; i < N; i++)
	{
		a[i] = i + 1;
		b[i] = -2;
	}

	// выделение памяти под массивы на GPU
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));

	// копирование данных в память GPU
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	Prod <<<1, N >>>(dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

	int prod = 0; //переменная для результата

	//подсчёт скалярного произведения
	for (int i = 0; i < N; i++)
	{
		prod += c[i];
	}

	//вывод результата
	printf("prod = %d\n", prod);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}