﻿#include "hip/hip_runtime.h"

#include <cmath>
#include <stdio.h>

#define N 100 // количество членов ряда

__global__ void ZFunction(float *a, float *b)
{
    int i = threadIdx.x; //индексация
    a[i] = 1.f / powf(float(i + 1), *b); //вычисление i-ого члена
}


int main()
{
    float s = 2; //степень
    float a[N]; //массив членов ряда
    float *d_s = 0;
    float *d_a = 0;
    float sum = 0; //частная сумма ряда

    hipMalloc((void**)&d_a, N * sizeof(float));
    hipMalloc((void**)&d_s, sizeof(float));
    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_s, &s, sizeof(float), hipMemcpyHostToDevice);

    ZFunction <<<1, N >>>(d_a, d_s);

    hipMemcpy(a, d_a, N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) 
    {
        sum += a[i];
    }
    printf("%f\n", sum);

    hipFree(d_a);
    hipFree(d_s);
    return 0;
}